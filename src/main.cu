#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <stdio.h>
#include <iostream>
#include <float.h>
#include <fstream>
#include <hiprand/hiprand_kernel.h>
#include "obj_loader.h"
#include "LocalRenderer/Window.h"
#include "LocalRenderer/Renderer.h"
#include "cuda_utils.h"
#include "Profiling/GPUMonitor.h"
#include "DevicePathTracer.h"
#include <chrono>
#include <cmath>
#include "SafeQueue.h"
#include "GPUThread.h"
#include "Scheduling/TaskGenerator.h"
#include <vector>

double getRadians(double value) {
    return M_PI * value / 180.0;
}

int main() {
    int view_width = 600;
    int view_height = 600;
    int num_pixels = view_width * view_height;
    size_t fb_size = num_pixels*sizeof(uint8_t) * 3;
    uint8_t *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // RenderTask task_0{800, 900, 0, 0};
    // RenderTask task_1{800, 900, 800, 0};

    // Load object
    const char *file_path = "models/cornell-box.obj";
    obj_loader loader(file_path);

    DevicePathTracer pt0(0, loader, view_width, view_height);
    DevicePathTracer pt1(1, loader, view_width, view_height);

    Window window(view_width, view_height, "MultiGPU-PathTracer");
    Renderer renderer(window);

    MonitorThread monitor_thread_obj;
    std::thread monitor_thread(std::ref(monitor_thread_obj));

    int x, y;
    bool firstMouse;
    double lastX, lastY;
    double yaw = 0, pitch = 0;

    // ----------------------------------------------------------------- //
    // SafeQueue<RenderTask> queue;
    // RenderTask task;
    // GPUThread t0(0, loader, view_width, view_height, queue, fb);
    // GPUThread t1(1, loader, view_width, view_height, queue, fb);
    // std::thread gpu_0_thread(std::ref(t0));
    // std::thread gpu_1_thread(std::ref(t1));
    // ----------------------------------------------------------------- //
    int num_streams_per_gpu = 4;
    TaskGenerator task_gen(view_width, view_height);

    std::vector<RenderTask> render_tasks;
    // task_gen.generateTasks(num_streams_per_gpu*2,render_tasks);
    task_gen.generateTasks(32,32,render_tasks);
    SafeQueue<RenderTask> queue;
    




    hipStream_t stream_0[num_streams_per_gpu];
    hipStream_t stream_1[num_streams_per_gpu];

    hipEvent_t event_0[num_streams_per_gpu];
    hipEvent_t event_1[num_streams_per_gpu];
    for (int i = 0; i < num_streams_per_gpu; i++) {
        hipSetDevice(0);
        hipStreamCreate(&stream_0[i]);
        hipEventCreate(&event_0[i]);

        hipSetDevice(1);
        hipStreamCreate(&stream_1[i]);
        hipEventCreate(&event_1[i]);
    }
    GPUThread t0_0(0,stream_0[0], loader, view_width, view_height, queue, fb);
    GPUThread t0_1(0,stream_0[1], loader, view_width, view_height, queue, fb);
    GPUThread t0_2(0,stream_0[2], loader, view_width, view_height, queue, fb);
    GPUThread t0_3(0,stream_0[3], loader, view_width, view_height, queue, fb);
    GPUThread t1_0(1,stream_1[0], loader, view_width, view_height, queue, fb);
    GPUThread t1_1(1,stream_1[1], loader, view_width, view_height, queue, fb);
    GPUThread t1_2(1,stream_1[2], loader, view_width, view_height, queue, fb);
    GPUThread t1_3(1,stream_1[3], loader, view_width, view_height, queue, fb);
    std::thread gpu_0_thread_0(std::ref(t0_0));
    std::thread gpu_0_thread_1(std::ref(t0_1));
    std::thread gpu_0_thread_2(std::ref(t0_2));
    std::thread gpu_0_thread_3(std::ref(t0_3));
    std::thread gpu_1_thread_0(std::ref(t1_0));
    // std::thread gpu_1_thread_1(std::ref(t1_1));
    // std::thread gpu_1_thread_2(std::ref(t1_2));
    // std::thread gpu_1_thread_3(std::ref(t1_3));


    

    while (!window.shouldClose()) {
        window.pollEvents();



        // insert elements
        for (int i = 0; i < render_tasks.size(); i++) {
            queue.Produce(std::move(render_tasks[i]));
        }
        
        window.getMousePos(x, y);

        if (firstMouse)
        {
            lastX = (double)x;
            lastY = (double)y;
            firstMouse = false;
        }

        double xoffset = (double)x - lastX;
        double yoffset = lastY - (double)y; 
        lastX = x;
        lastY = y;

        double sensitivity = 0.5f;
        xoffset *= sensitivity;
        yoffset *= sensitivity;

        yaw += xoffset;
        pitch += yoffset;

        if (pitch > 89.0f)
            pitch = 89.0f;
        if (pitch < -89.0f)
            pitch = -89.0f;

        float3 lookat = make_float3(cos(getRadians(yaw)) * cos(getRadians(pitch)), 
                                   sin(getRadians(pitch)), 
                                   sin(getRadians(yaw)) * cos(getRadians(pitch)));

        pt0.setLookAt(lookat);
        pt1.setLookAt(lookat);

        auto start = std::chrono::high_resolution_clock::now();
        // t0_0.devicePathTracer.setLookAt(lookat);

        // for (int i = 0; i < num_streams_per_gpu; i++) {
        //     pt0.renderTaskAsync(render_tasks[i], fb, stream_0[i]);
        //     pt1.renderTaskAsync(render_tasks[i + num_streams_per_gpu], fb, stream_1[i]);
        // }
        
        // for (int i = 0; i < num_streams_per_gpu; i++) {
        //     hipEventRecord(event_0[i], stream_0[i]);
        //     hipEventRecord(event_1[i], stream_1[i]);
        // }
        
        // for(int i = 0; i < num_streams_per_gpu; i++) {
        //     hipEventSynchronize(event_0[i]);
        //     hipEventSynchronize(event_1[i]);
        // }
        

        auto stop = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
        std::cout << "path tracing took: " << duration.count() << "ms" << std::endl;

        renderer.renderFrame(fb);
	    window.swapBuffers();	
	}

    monitor_thread_obj.safeTerminate();
    monitor_thread.join();

    checkCudaErrors(hipFree(fb));
    return 0;
}
