#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <stdio.h>
#include <iostream>
#include <float.h>
#include <fstream>
#include <hiprand/hiprand_kernel.h>
#include "semaphore.h"
#include <mutex>
#include "obj_loader.h"
#include "LocalRenderer/Window.h"
#include "LocalRenderer/Renderer.h"
#include "cuda_utils.h"
#include "Profiling/GPUMonitor.h"
#include "DevicePathTracer.h"
#include <chrono>
#include <ixwebsocket/IXNetSystem.h>
#include <ixwebsocket/IXWebSocket.h>
#include <ixwebsocket/IXUserAgent.h>
#include <cmath>
#include "SafeQueue.h"
#include "GPUThread.h"
#include "hip/hip_vector_types.h"
#include "CameraParams.h"
#include "Scheduling/TaskGenerator.h"
#include <vector>

double getRadians(double value) {
    return M_PI * value / 180.0;
}

int main(int argc, char **argv) {
    int view_width = 600;
    int view_height = 600;
    int num_pixels = view_width * view_height;
    size_t fb_size = num_pixels*sizeof(uint8_t) * 3;
    uint8_t *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // Load object
    char *file_path;
    if (argc > 2) {
        file_path = argv[2];
    } else {
        file_path = "models/cubes.obj";
    }
    obj_loader loader(file_path);

    // Load job id
    char *job_id;
    if (argc > 1) {
        job_id = argv[1];
    } else {
        job_id = "0";
    }

    ix::WebSocket webSocket;
    std::string url("wss://pathtracing-relay-server.klatka.it");
    webSocket.setUrl(url);

    // // Setup a callback to be fired (in a background thread, watch out for race conditions !)
    // // when a message or an event (open, close, error) is received
    webSocket.setOnMessageCallback([](const ix::WebSocketMessagePtr& msg)
        {
            if (msg->type == ix::WebSocketMessageType::Message)
            {
                std::cout << "received message: " << msg->str << std::endl;
            }
            else if (msg->type == ix::WebSocketMessageType::Open)
            {
                std::cout << "Connection established" << std::endl;
            }
            else if (msg->type == ix::WebSocketMessageType::Error)
            {
                // Maybe SSL is not configured properly
                std::cout << "Connection error: " << msg->errorInfo.reason << std::endl;
            }
        }
    );

    // Now that our callback is setup, we can start our background thread and receive messages
    webSocket.start();

    // DevicePathTracer pt0(0, loader, view_width, view_height);
    // DevicePathTracer pt1(1, loader, view_width, view_height);
    CameraParams camParams;
    camParams.lookFrom = make_float3(-277.676, 157.279, 545.674);
    camParams.front = make_float3(-0.26, 0.121, -0.9922);

    Window window(view_width, view_height, "MultiGPU-PathTracer", camParams);
    Renderer renderer(window);

    MonitorThread monitor_thread_obj;
    std::thread monitor_thread(std::ref(monitor_thread_obj));

    // ----------------------------------------------------------------- //
    // SafeQueue<RenderTask> queue;
    // RenderTask task;
    // GPUThread t0(0, loader, view_width, view_height, queue, fb);
    // GPUThread t1(1, loader, view_width, view_height, queue, fb);
    // std::thread gpu_0_thread(std::ref(t0));
    // std::thread gpu_1_thread(std::ref(t1));
    // ----------------------------------------------------------------- //
    int num_streams_per_gpu = 4;
    TaskGenerator task_gen(view_width, view_height);

    std::vector<RenderTask> render_tasks;

    task_gen.generateTasks(32,32,render_tasks);
    SafeQueue<RenderTask> queue;
    
    std::condition_variable thread_cv;
    semaphore thread_semaphore(0);
    std::atomic_int completed_streams = 0;



    hipStream_t stream_0[num_streams_per_gpu];
    hipStream_t stream_1[num_streams_per_gpu];

    hipEvent_t event_0[num_streams_per_gpu];
    hipEvent_t event_1[num_streams_per_gpu];
    for (int i = 0; i < num_streams_per_gpu; i++) {
        hipSetDevice(0);
        hipStreamCreate(&stream_0[i]);
        hipEventCreate(&event_0[i]);

        hipSetDevice(1);
        hipStreamCreate(&stream_1[i]);
        hipEventCreate(&event_1[i]);
    }
    GPUThread t0_0(0,stream_0[0], loader, view_width, view_height, queue, fb, &thread_semaphore, &thread_cv, &completed_streams, camParams);
    GPUThread t0_1(0,stream_0[1], loader, view_width, view_height, queue, fb, &thread_semaphore, &thread_cv, &completed_streams, camParams);
    GPUThread t0_2(0,stream_0[2], loader, view_width, view_height, queue, fb, &thread_semaphore, &thread_cv, &completed_streams, camParams);
    GPUThread t0_3(0,stream_0[3], loader, view_width, view_height, queue, fb, &thread_semaphore, &thread_cv, &completed_streams, camParams);
    GPUThread t1_0(1,stream_1[0], loader, view_width, view_height, queue, fb, &thread_semaphore, &thread_cv, &completed_streams, camParams);
    GPUThread t1_1(1,stream_1[1], loader, view_width, view_height, queue, fb, &thread_semaphore, &thread_cv, &completed_streams, camParams);
    GPUThread t1_2(1,stream_1[2], loader, view_width, view_height, queue, fb, &thread_semaphore, &thread_cv, &completed_streams, camParams);
    GPUThread t1_3(1,stream_1[3], loader, view_width, view_height, queue, fb, &thread_semaphore, &thread_cv, &completed_streams, camParams);
    std::thread gpu_0_thread_0(std::ref(t0_0));
    std::thread gpu_0_thread_1(std::ref(t0_1));
    std::thread gpu_0_thread_2(std::ref(t0_2));
    std::thread gpu_0_thread_3(std::ref(t0_3));
    std::thread gpu_1_thread_0(std::ref(t1_0));
    std::thread gpu_1_thread_1(std::ref(t1_1));
    std::thread gpu_1_thread_2(std::ref(t1_2));
    std::thread gpu_1_thread_3(std::ref(t1_3));

    std::mutex m;
    std::unique_lock<std::mutex> lk(m);

    

    while (!window.shouldClose()) {
        window.pollEvents();
        // pt0.setFront(camParams.front);
        // pt0.setLookFrom(camParams.lookFrom);

        // pt1.setFront(camParams.front);
        // pt1.setLookFrom(camParams.lookFrom);

         // insert elements
        for (int i = 0; i < render_tasks.size(); i++) {
            queue.Produce(std::move(render_tasks[i]));
        }

        auto start = std::chrono::high_resolution_clock::now();

        thread_semaphore.release(2*num_streams_per_gpu);
        while(completed_streams != num_streams_per_gpu * 2) {
            thread_cv.wait(lk);
        }
        completed_streams = 0;

        auto stop = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
        // std::cout << "path tracing took: " << duration.count() << "ms" << std::endl;

        std::string pixel_data = "";
        for (int j = view_height-1; j >= 0; j--) {
            for (int i = 0; i < view_width; i++) {
                size_t pixel_index = j*view_width + i;
                auto x = fb[3*pixel_index];
                auto y = fb[3*pixel_index+1];
                auto z = fb[3*pixel_index+2];
                pixel_data += std::to_string(x) + "," + std::to_string(y) + "," + std::to_string(z) + ",";
            }
        }

        // Send pixel data to server
        webSocket.send("JOB_MESSAGE#" + std::string(job_id) + "#" + pixel_data);

        // renderer.renderFrame(fb);
	    // window.swapBuffers();	
	}

    monitor_thread_obj.safeTerminate();
    monitor_thread.join();

    checkCudaErrors(hipFree(fb));
    return 0;
}
