#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <float.h>
#include <fstream>
#include <hiprand/hiprand_kernel.h>
#include "ray.h"
#include "hip/hip_vector_types.h"
#include "hitable_list.h"
#include "sphere.h"
#include "hitable.h"
#include "camera.h"
#include "material.h"
#include "bvh.h"







#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA ERROR = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        const char* error_string = hipGetErrorString(result); 
        std::cerr << error_string << " " << std::endl;
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}


/**
 * Initializes the rendering process.
 *
 * This kernel function is responsible for initializing the rendering process by setting up the random number generator states.
 *
 * @param nx The width of the image.
 * @param ny The height of the image.
 * @param rand_state Pointer to the array of random number generator states.
 */
__global__ void render_init(int nx, int ny, hiprandState *rand_state) {
    //render_init doesnt have to be separate kernel, dona that way for clarity 
    //better performance to do it in the render kernel (will change later)
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= nx) || (j >= ny)) return;
    int pixel_index = j*nx + i;
    //Each thread gets diffrent seed, same sequence number, no offset
    hiprand_init(1984+pixel_index, 0, 0, &rand_state[pixel_index]);
}

/**
 * @brief Renders the scene using path tracing algorithm.
 *
 * This CUDA kernel function is responsible for rendering the scene using the path tracing algorithm.
 * It takes in the framebuffer `fb`, the maximum width and height of the image `max_x` and `max_y`,
 * the number of samples per pixel `sample_per_pixel`, an array of camera pointers `cam`, an array of
 * hitable pointers `world`, and the random state for each thread `rand_state`.
 *
 * @param fb The framebuffer to store the rendered image.
 * @param max_x The maximum width of the image.
 * @param max_y The maximum height of the image.
 * @param sample_per_pixel The number of samples per pixel.
 * @param cam An array of camera pointers.
 * @param world An array of hitable pointers representing the scene.
 * @param rand_state The random state for each thread.
 */
__global__ void render(float3 *fb, int max_x, int max_y,int sample_per_pixel, camera **cam,bvh **world, hiprandState *rand_state) {
    
   int i = threadIdx.x + blockIdx.x * blockDim.x;
   int j = threadIdx.y + blockIdx.y * blockDim.y;
   if((i >= max_x) || (j >= max_y)) return;
   int pixel_index = j*max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    //Antialiasing
    float3 col = make_float3(0, 0, 0);
    for (int s=0; s<sample_per_pixel; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v);
        col += (*cam)->ray_color(r, world, &local_rand_state);
    }

   fb[pixel_index] = col/float(sample_per_pixel); //average color of samples
}

/**
 * @brief CUDA kernel to create the world, list of objects, and camera.
 *
 * This CUDA kernel is responsible for creating the world, list of objects, and camera
 * in the GPU memory. It takes in pointers to the device memory where the list of objects,
 * world, and camera will be stored.
 *
 * @param d_list Pointer to the device memory where the list of objects will be stored.
 * @param d_world Pointer to the device memory where the world will be stored.
 * @param d_camera Pointer to the device memory where the camera will be stored.
 */
__global__ void create_world(hitable **d_list, bvh **d_world,camera **d_camera) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        d_list[0] = new sphere(make_float3(0,0,-1), 0.5,
                               new lambertian(make_float3(0.7, 0.7, 0.5)));
        d_list[1] = new sphere(make_float3(0,-100.5,-1), 100,
                               new lambertian(make_float3(0.8, 0.8, 0.0)));
        d_list[2] = new sphere(make_float3(1,0,-1), 0.5,
                               new metal(make_float3(0.8, 0.6, 0.8),0.0));
        d_list[3] = new sphere(make_float3(-1,0,-1), 0.5, //negative radius trick makes it face inwards
                               new dielectric(1.5f));
        d_list[4] = new sphere(make_float3(-1,0,-1), -0.45,
                                 new dielectric(1.5));                    
        *d_world  = new bvh(d_list,5);
        *d_camera = new camera();
    }
}

__global__ void free_world(hitable **d_list, bvh **d_world,camera **d_camera) {
    for(int i=0; i < 5; i++) {
        delete ((sphere *)d_list[i])->mat_ptr;
        delete d_list[i];
    }
    delete *d_world;
    delete *d_camera;
}

int main()
{
    int nx = 1600;
    int ny = 900;
    
    int tx = 8; //thread amount should be a multiple of 32
    int ty = 8;
    float aspect_ratio = float(nx) / float(ny);

    int num_pixels = nx*ny;
    size_t fb_size = num_pixels*sizeof(float3);

    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);

    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));
    // allocate FB
    float3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    //create_world
    hitable **d_list;
    checkCudaErrors(hipMalloc((void **)&d_list, 5*sizeof(hitable *)));
    bvh **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(bvh *)));
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    create_world<<<1,1>>>(d_list,d_world,d_camera);
    
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    
    //render
    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    // return;
    render<<<blocks, threads>>>(fb, nx, ny,
        100, d_camera,
        d_world,
        d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    // free_world
    free_world<<<1, 1>>>(d_list, d_world,d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    //output
    std::ofstream myfile;
    myfile.open("out.ppm");
    myfile << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*nx + i;
            int3 color = make_int3(255.99*fb[pixel_index].x, 255.99*fb[pixel_index].y, 255.99*fb[pixel_index].z);
            myfile << color.x << " " << color.y << " " << color.z << "\n";
        }
    }
    myfile.close();
    checkCudaErrors(hipFree(fb));

    return 0;
}
