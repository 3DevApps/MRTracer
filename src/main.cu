#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <stdio.h>
#include <iostream>
#include <float.h>
#include <fstream>
#include <hiprand/hiprand_kernel.h>
#include "semaphore.h"
#include <mutex>
#include "obj_loader.h"
#include "LocalRenderer/Window.h"
#include "LocalRenderer/Renderer.h"
#include "cuda_utils.h"
#include "Profiling/GPUMonitor.h"
#include "DevicePathTracer.h"
#include <chrono>
#include <cmath>
#include "SafeQueue.h"
#include "GPUThread.h"
#include "hip/hip_vector_types.h"
#include "CameraParams.h"
#include "Scheduling/TaskGenerator.h"
#include <vector>

double getRadians(double value) {
    return M_PI * value / 180.0;
}

int main() {
    int view_width = 600;
    int view_height = 600;
    int num_pixels = view_width * view_height;
    size_t fb_size = num_pixels*sizeof(uint8_t) * 3;
    uint8_t *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // Load object
    const char *file_path = "models/cornell-box.obj";
    obj_loader loader(file_path);

    DevicePathTracer pt0(0, loader, view_width, view_height);
    DevicePathTracer pt1(1, loader, view_width, view_height);
    CameraParams camParams;
    camParams.lookFrom = make_float3(-277.676, 157.279, 545.674);
    camParams.front = make_float3(-0.26, 0.121, -0.9922);

    Window window(view_width, view_height, "MultiGPU-PathTracer", camParams);
    Renderer renderer(window);

    MonitorThread monitor_thread_obj;
    std::thread monitor_thread(std::ref(monitor_thread_obj));

    int x, y;
    bool firstMouse;
    double lastX, lastY;
    double yaw = 0, pitch = 0;

    // ----------------------------------------------------------------- //
    // SafeQueue<RenderTask> queue;
    // RenderTask task;
    // GPUThread t0(0, loader, view_width, view_height, queue, fb);
    // GPUThread t1(1, loader, view_width, view_height, queue, fb);
    // std::thread gpu_0_thread(std::ref(t0));
    // std::thread gpu_1_thread(std::ref(t1));
    // ----------------------------------------------------------------- //
    int num_streams_per_gpu = 4;
    TaskGenerator task_gen(view_width, view_height);

    std::vector<RenderTask> render_tasks;

    task_gen.generateTasks(32,32,render_tasks);
    SafeQueue<RenderTask> queue;
    
    std::condition_variable thread_cv;
    semaphore thread_semaphore(0);
    std::atomic_int completed_streams = 0;



    hipStream_t stream_0[num_streams_per_gpu];
    hipStream_t stream_1[num_streams_per_gpu];

    hipEvent_t event_0[num_streams_per_gpu];
    hipEvent_t event_1[num_streams_per_gpu];
    for (int i = 0; i < num_streams_per_gpu; i++) {
        hipSetDevice(0);
        hipStreamCreate(&stream_0[i]);
        hipEventCreate(&event_0[i]);

        hipSetDevice(1);
        hipStreamCreate(&stream_1[i]);
        hipEventCreate(&event_1[i]);
    }
    GPUThread t0_0(0,stream_0[0], loader, view_width, view_height, queue, fb, &thread_semaphore, &thread_cv, &completed_streams);
    GPUThread t0_1(0,stream_0[1], loader, view_width, view_height, queue, fb, &thread_semaphore, &thread_cv, &completed_streams);
    GPUThread t0_2(0,stream_0[2], loader, view_width, view_height, queue, fb, &thread_semaphore, &thread_cv, &completed_streams);
    GPUThread t0_3(0,stream_0[3], loader, view_width, view_height, queue, fb, &thread_semaphore, &thread_cv, &completed_streams);
    GPUThread t1_0(1,stream_1[0], loader, view_width, view_height, queue, fb, &thread_semaphore, &thread_cv, &completed_streams);
    GPUThread t1_1(1,stream_1[1], loader, view_width, view_height, queue, fb, &thread_semaphore, &thread_cv, &completed_streams);
    GPUThread t1_2(1,stream_1[2], loader, view_width, view_height, queue, fb, &thread_semaphore, &thread_cv, &completed_streams);
    GPUThread t1_3(1,stream_1[3], loader, view_width, view_height, queue, fb, &thread_semaphore, &thread_cv, &completed_streams);
    std::thread gpu_0_thread_0(std::ref(t0_0));
    std::thread gpu_0_thread_1(std::ref(t0_1));
    std::thread gpu_0_thread_2(std::ref(t0_2));
    std::thread gpu_0_thread_3(std::ref(t0_3));
    std::thread gpu_1_thread_0(std::ref(t1_0));
    std::thread gpu_1_thread_1(std::ref(t1_1));
    std::thread gpu_1_thread_2(std::ref(t1_2));
    std::thread gpu_1_thread_3(std::ref(t1_3));

    std::mutex m;
    std::unique_lock<std::mutex> lk(m);

    

    while (!window.shouldClose()) {
        window.pollEvents();


        pt0.setFront(camParams.front);
        pt0.setLookFrom(camParams.lookFrom);

        pt1.setFront(camParams.front);
        pt1.setLookFrom(camParams.lookFrom);

         // insert elements
        for (int i = 0; i < render_tasks.size(); i++) {
            queue.Produce(std::move(render_tasks[i]));
        }

        auto start = std::chrono::high_resolution_clock::now();

        thread_semaphore.release(2*num_streams_per_gpu);
        while(completed_streams != num_streams_per_gpu * 2) {
            thread_cv.wait(lk);
        }
        completed_streams = 0;

        auto stop = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
        // std::cout << "path tracing took: " << duration.count() << "ms" << std::endl;

        renderer.renderFrame(fb);
	    window.swapBuffers();	
	}

    monitor_thread_obj.safeTerminate();
    monitor_thread.join();

    checkCudaErrors(hipFree(fb));
    return 0;
}
