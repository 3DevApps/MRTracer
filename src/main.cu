#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <float.h>
#include <fstream>
#include <hiprand/hiprand_kernel.h>
#include "ray.h"
#include "hip/hip_vector_types.h"
#include "hitable_list.h"
#include "sphere.h"
#include "hitable.h"
#include "camera.h"
#include "material.h"



#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

//render_init doesnt have to be separate kernel, dona that way for clarity 
//better performance to do it in the render kernel
__global__ void render_init(int nx, int ny, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= nx) || (j >= ny)) return;
    int pixel_index = j*nx + i;
    //Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(float3 *fb, int max_x, int max_y,int sample_per_pixel, camera **cam,hitable **world, hiprandState *rand_state) {
   int i = threadIdx.x + blockIdx.x * blockDim.x;
   int j = threadIdx.y + blockIdx.y * blockDim.y;
   if((i >= max_x) || (j >= max_y)) return;
   int pixel_index = j*max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    float3 col = make_float3(0, 0, 0);
    for (int s=0; s<sample_per_pixel; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v);
        col += (*cam)->ray_color(r, world, &local_rand_state);
    }

   fb[pixel_index] = col/float(sample_per_pixel);
}

__global__ void create_world(hitable **d_list, hitable **d_world,camera **d_camera) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        d_list[0] = new sphere(make_float3(0,0,-0.7), 0.5,
                               new lambertian(make_float3(0.8, 0.3, 0.3)));
        d_list[1] = new sphere(make_float3(0,-100.5,-0.7), 100,
                               new lambertian(make_float3(0.8, 0.8, 0.0)));
        d_list[2] = new sphere(make_float3(1,0,-0.7), 0.5,
                               new metal(make_float3(0.8, 0.6, 0.2),1.0));
        d_list[3] = new sphere(make_float3(-1,0,-0.7), 0.5,
                               new metal(make_float3(0.8, 0.8, 0.8),0.3));
        *d_world  = new hitable_list(d_list,4);
        *d_camera = new camera();
    }
}

__global__ void free_world(hitable **d_list, hitable **d_world,camera **d_camera) {
    delete *(d_list);
    delete *(d_list+1);
    delete *d_world;
    delete *d_camera;
    
}

int main()
{
    int nx = 1600;
    int ny = 900;
    
    int tx = 8; //thread amount should be a multiple of 32
    int ty = 8;
    float aspect_ratio = float(nx) / float(ny);

    int num_pixels = nx*ny;
    size_t fb_size = num_pixels*sizeof(float3);

    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);

    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));
    // allocate FB
    float3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    //create_world
    hitable **d_list;
    checkCudaErrors(hipMalloc((void **)&d_list, 4*sizeof(hitable *)));
    hitable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    create_world<<<1,1>>>(d_list,d_world,d_camera);
    
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    //render
    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads>>>(fb, nx, ny,
        100, d_camera,
        d_world,
        d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    //free_world
    free_world<<<1, 1>>>(d_list, d_world,d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    //output
    std::ofstream myfile;
    myfile.open("out.ppm");
    myfile << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*nx + i;
            int3 color = make_int3(255.99*fb[pixel_index].x, 255.99*fb[pixel_index].y, 255.99*fb[pixel_index].z);
            myfile << color.x << " " << color.y << " " << color.z << "\n";
        }
    }
    myfile.close();
    checkCudaErrors(hipFree(fb));

    return 0;
}