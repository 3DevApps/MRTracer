#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <stdio.h>
#include <iostream>
#include <float.h>
#include <fstream>
// #include <hiprand/hiprand_kernel.h>
#include "semaphore.h"
#include <mutex>
#include "Renderer/LocalRenderer/Window.h"
#include "Renderer/LocalRenderer/LocalRenderer.h"
// #include "cuda_utils.h"
#include "Profiling/GPUMonitor.h"
#include "DevicePathTracer.h"
#include <chrono>
#include <cmath>
#include "SafeQueue.h"
#include "StreamThread.h"
// #include "hip/hip_vector_types.h"
#include "HostScene.h"
#include "Scheduling/TaskGenerator.h"
#include <vector>
#include "PixelDataEncoder/PixelDataEncoder.h"
#include "PixelDataEncoder/JPEGEncoder.h"
#include "PixelDataEncoder/PNGEncoder.h"
#include "ArgumentLoader.h"
#include "Renderer/RemoteRenderer/RemoteRenderer.h"
#include "Renderer/Renderer.h"
#include "Renderer/RemoteRenderer/RemoteEventHandlers/RemoteEventHandlers.h"
#include "RendererConfig.h"
#include "Framebuffer.h"
#include "RenderManager.h"

int main(int argc, char** argv) {
    ArgumentLoader argLoader(argc, argv);
    auto args = argLoader.loadAndGetArguments();

    RendererConfig config; 
    HostScene hScene(config.objPath, make_float3(0.05, 0.05, 0.05), make_float3(-0.05, -0.05, -0.05));
    Window window(config.resolution.width, config.resolution.height, "MultiGPU-PathTracer", hScene.cameraParams);
    RenderManager manager(config, hScene);
    /*
    changing parameters:
    manager.setSamplesPerPixel(30);
    manager.setRecursionDepth(5);
    manager.setGpuAndStreamNumber(1, 6);
    manager.setResolution({900, 900}); // TODO: make rendered frame resolution independent from window size
    manager.setThreadBlockSize({16, 16});

    hScene.loadTriangles("path/to/obj");
    hScene.setVFOV(60.0f);
    hScene.setHFOV(60.0f);
    hScene.setCameraLookFrom(make_float3(1, 1, 1));
    hScene.setCameraFront(make_float3(1, 1, 1));
    */

    LocalRenderer localRenderer(window);
    RemoteRenderer remoteRenderer(args.jobId, config.resolution.width, config.resolution.height);
    RemoteEventHandlers remoteEventHandlers(remoteRenderer, hScene.cameraParams);
    Renderer &renderer = localRenderer;

    MonitorThread monitor_thread_obj(renderer);
    std::thread monitor_thread(std::ref(monitor_thread_obj));

    while (!renderer.shouldStopRendering()) {
        
        auto start = std::chrono::high_resolution_clock::now();
        manager.renderFrame();
        auto stop = std::chrono::high_resolution_clock::now();

        auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
        std::cout << "path tracing took: " << duration.count() << "ms" << std::endl;

        renderer.renderFrame(manager.getCurrentFrame());
	    window.swapBuffers(); 	        
	}

    manager.reset();
    monitor_thread_obj.safeTerminate();
    monitor_thread.join();
    return 0;
}