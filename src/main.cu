#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <float.h>
#include <fstream>
#include <hiprand/hiprand_kernel.h>
#include "ray.h"
#include "hip/hip_vector_types.h"
#include "hitable_list.h"
#include "sphere.h"
#include "hitable.h"
#include "camera.h"
#include "material.h"
#include "obj_loader.h"
#include "object3d.h"
#include "triangle.h"



#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA ERROR = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

//render_init doesnt have to be separate kernel, dona that way for clarity 
//better performance to do it in the render kernel
__global__ void render_init(int nx, int ny, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= nx) || (j >= ny)) return;
    int pixel_index = j*nx + i;
    //Each thread gets diffrent seed, same sequence number, no offset
    hiprand_init(1984+pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void render(float3 *fb, int max_x, int max_y,int sample_per_pixel, camera **cam,hitable **world, hiprandState *rand_state) {
   int i = threadIdx.x + blockIdx.x * blockDim.x;
   int j = threadIdx.y + blockIdx.y * blockDim.y;
   if((i >= max_x) || (j >= max_y)) return;
   int pixel_index = j*max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    float3 col = make_float3(0, 0, 0);
    for (int s=0; s<sample_per_pixel; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v);
        col += (*cam)->ray_color(r, world, &local_rand_state);
    }

   fb[pixel_index] = col/float(sample_per_pixel);
}

__global__ void create_world(hitable **d_list, hitable **d_world,camera **d_camera, triangle *triangles, int num_triangles) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("Creating world %d\n", num_triangles);
        
        for (int i = 0; i < num_triangles; i++) {
            d_list[i] = new triangle(triangles[i].v0, triangles[i].v1, triangles[i].v2, new metal(make_float3(0.8, 0.6, 0.2),0.0));
        }

        // d_list[0] = new object3d(triangles, num_triangles, new metal(make_float3(0.8, 0.6, 0.2),0.0));

        // obj->set_material(new metal(make_float3(0.8, 0.6, 0.2),0.0));

        // object3d obj1 = obj->clone();
        // obj1.set_material(new metal(make_float3(0.8, 0.6, 0.2),0.0));

        // d_list[0] = &obj1;
        // d_list[1] = &obj1;
        // d_list[2] = &obj1;
        // d_list[3] = &obj1;
        // d_list[4] = &obj1;

        // d_list[0] = new object3d(obj->triangles, obj->num_triangles, obj->mat_ptr);
        // d_list[1] = obj->clone();
        // d_list[2] = obj->clone();
        // d_list[3] = obj->clone();
        // d_list[4] = obj->clone();

        // printf("num_triangles: %d\n", obj->num_triangles);

        // printf("triangle %f\n", obj->triangles[15].v0.x);
        // d_list[0] = new object3d(obj->triangles, obj->num_triangles, new metal(make_float3(0.8, 0.6, 0.2),0.0));
        // d_list[1] = new object3d(obj->triangles, obj->num_triangles, new metal(make_float3(0.8, 0.6, 0.2),0.0));
        // d_list[2] = new object3d(obj->triangles, obj->num_triangles, new metal(make_float3(0.8, 0.6, 0.2),0.0));
        // d_list[3] = new object3d(obj->triangles, obj->num_triangles, new metal(make_float3(0.8, 0.6, 0.2),0.0));
        // d_list[4] = new object3d(obj->triangles, obj->num_triangles, new metal(make_float3(0.8, 0.6, 0.2),0.0));


        // d_list[0]->print_gpu();


        // d_list[1] = new object3d(new metal(make_float3(0.8, 0.6, 0.2),0.0));
        // d_list[2] = new object3d( new metal(make_float3(0.8, 0.6, 0.2),0.0));
        // d_list[3] = new object3d( new metal(make_float3(0.8, 0.6, 0.2),0.0));
        // d_list[4] = new object3d( new metal(make_float3(0.8, 0.6, 0.2),0.0));

        // float3 v0 = make_float3(1,0,-1);
        // float3 v1 = make_float3(2,0,-1);
        // float3 v2 = make_float3(0,1,-1);

        // d_list[0] = new triangle(v0, v1, v2, new metal(make_float3(0.8, 0.6, 0.2),0.0));
        // d_list[1] = new sphere(make_float3(0,-100.5,-1), 100,
        //                        new lambertian(make_float3(0.8, 0.8, 0.0)));
        // d_list[2] = new sphere(make_float3(1,0,-1), 0.5,
        //                          new metal(make_float3(0.8, 0.6, 0.2),0.0));
        // d_list[3] = new sphere(make_float3(-1,0,-1), 0.5, //negative radius trick makes it face inwards
        //                             new dielectric(1.5f));
        // d_list[4] = new sphere(make_float3(-1,0,-1), -0.45,
        //                             new dielectric(1.5));
        // d_list[0] = new triangle(v0, v1, v2, new metal(make_float3(0.8, 0.6, 0.2),0.0));
        // d_list[2] = new triangle(v0, v1, v2, new metal(make_float3(0.8, 0.6, 0.2),0.0));
        // d_list[3] = new triangle(v0, v1, v2, new metal(make_float3(0.8, 0.6, 0.2),0.0));
        // d_list[4] = new triangle(v0, v1, v2, new metal(make_float3(0.8, 0.6, 0.2),0.0));


        // d_list[0]->print_gpu();

        // d_list[0] = new sphere(make_float3(0,-100.5,-1), 100,
                            //    new lambertian(make_float3(0.8, 0.8, 0.0)));
        // d_list[1] = new sphere(make_float3(0,-100.5,-1), 100,
        //                        new lambertian(make_float3(0.8, 0.8, 0.0)));
        // d_list[2] = new sphere(make_float3(1,0,-1), 0.5,
        //                        new metal(make_float3(0.8, 0.6, 0.2),0.0));
        // d_list[3] = new sphere(make_float3(-1,0,-1), 0.5, //negative radius trick makes it face inwards
        //                        new dielectric(1.5f));
        // d_list[4] = new sphere(make_float3(-1,0,-1), -0.45,
        //                          new dielectric(1.5));                    
        *d_world  = new hitable_list(d_list, num_triangles);
        *d_camera = new camera();
    }
}

__global__ void free_world(hitable **d_list, hitable **d_world,camera **d_camera) {
    for(int i=0; i < 5; i++) {
        // delete ((sphere *)d_list[i])->mat_ptr;
        delete d_list[i];
    }
    delete *d_world;
    delete *d_camera;
    
}

int main()
{
    int nx = 1600;
    int ny = 900;
    // int nx = 1200;
    // int ny = 600;
    
    int tx = 8; //thread amount should be a multiple of 32
    int ty = 8;
    float aspect_ratio = float(nx) / float(ny);

    int num_pixels = nx*ny;
    size_t fb_size = num_pixels*sizeof(float3);

    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);

    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));
    // allocate FB
    float3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    //create_world
    // Load object
    // TODO: this should be a parameter to the program
    object3d *obj;
    checkCudaErrors(hipMallocManaged((void **)&obj, sizeof(object3d)));

    obj_loader loader;
    loader.load(obj, "models/cubes.obj");

    // Create array of triangles from object
    // TODO: this should be done in the obj_loader
    triangle *triangles;
    checkCudaErrors(hipMallocManaged((void **)&triangles, obj->num_triangles * sizeof(triangle)));

    for (int i = 0; i < obj->num_triangles; i++) {
        triangles[i] = triangle(obj->triangles[i].v0, obj->triangles[i].v1, obj->triangles[i].v2);
    }

    hitable **d_list;
    checkCudaErrors(hipMalloc((void **)&d_list, obj->num_triangles*sizeof(hitable *)));
    hitable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));

    
    // TODO: we should pass the object_3d and not triangles array
    create_world<<<1,1>>>(d_list,d_world,d_camera,triangles, obj->num_triangles);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    //render
    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads>>>(fb, nx, ny,
        100, d_camera,
        d_world,
        d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    //free_world
    free_world<<<1, 1>>>(d_list, d_world,d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    //output
    std::ofstream myfile;
    myfile.open("out.ppm");
    myfile << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*nx + i;
            int3 color = make_int3(255.99*fb[pixel_index].x, 255.99*fb[pixel_index].y, 255.99*fb[pixel_index].z);
            myfile << color.x << " " << color.y << " " << color.z << "\n";
        }
    }
    myfile.close();
    checkCudaErrors(hipFree(fb));

    return 0;
}
